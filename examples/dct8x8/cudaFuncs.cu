#include <hip/hip_runtime.h>
#include "BmpUtil.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>       // helper functions for CUDA timing and initialization
#include <helper_functions.h>  // helper functions for timing, string parsing

#define BENCHMARK_SIZE  10

texture<float, 2, hipReadModeElementType> TexSrc;

#include "dct8x8_kernel1.cuh"
#include "dct8x8_kernel_quantization.cuh"
#include "dct8x8_kernel2.cuh"


namespace dct8x8ModuleKernels{

  float WrapperCUDA1(byte *ImgSrc, byte *ImgDst, int Stride, ROI Size){
    hipChannelFormatDesc floattex = hipCreateChannelDesc<float>();

    //allocate device memory
    hipArray *Src;
    float *Dst;
    size_t DstStride;
    //printf("Allocating memory\n");
    checkCudaErrors(hipMallocArray(&Src, &floattex, Size.width, Size.height));
    checkCudaErrors(hipMallocPitch((void **)(&Dst), &DstStride, Size.width * sizeof(float), Size.height));
    DstStride /= sizeof(float);
    //printf("Converting img format\n");
    //convert source image to float representation
    int ImgSrcFStride;
    float *ImgSrcF = MallocPlaneFloat(Size.width, Size.height, &ImgSrcFStride);
    CopyByte2Float(ImgSrc, Stride, ImgSrcF, ImgSrcFStride, Size);
    AddFloatPlane(-128.0f, ImgSrcF, ImgSrcFStride, Size);

    //copy from host memory to device
    //printf("Copy to 2d array\n");
    checkCudaErrors(hipMemcpy2DToArray(Src, 0, 0,
                                        ImgSrcF, ImgSrcFStride * sizeof(float),
                                        Size.width * sizeof(float), Size.height,
                                        hipMemcpyHostToDevice));

    //setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(Size.width / BLOCK_SIZE, Size.height / BLOCK_SIZE);

    //create and start CUDA timer
    StopWatchInterface *timerCUDA = 0;
    sdkCreateTimer(&timerCUDA);
    sdkResetTimer(&timerCUDA);
    //printf("Bind texture\n");
    //execute DCT kernel and benchmark
    checkCudaErrors(hipBindTextureToArray(TexSrc, Src));
    //printf("Run kernel\n");
    for (int i=0; i<BENCHMARK_SIZE; i++)
      {
        sdkStartTimer(&timerCUDA);
        CUDAkernel1DCT<<< grid, threads >>>(Dst, (int) DstStride, 0, 0);
        checkCudaErrors(hipDeviceSynchronize());
        sdkStopTimer(&timerCUDA);
      }
    //printf("Unbind kernel\n");
    checkCudaErrors(hipUnbindTexture(TexSrc));
    getLastCudaError("Kernel execution failed");

    // finalize CUDA timer
    float TimerCUDASpan = sdkGetAverageTimerValue(&timerCUDA);
    sdkDeleteTimer(&timerCUDA);
    //printf("Run Quantization\n");
    // execute Quantization kernel
    CUDAkernelQuantizationFloat<<< grid, threads >>>(Dst, (int) DstStride);
    getLastCudaError("Kernel execution failed");
    //printf("Copy Quantization\n");
    //copy quantized coefficients from host memory to device array
    checkCudaErrors(hipMemcpy2DToArray(Src, 0, 0,
                                        Dst, DstStride *sizeof(float),
                                        Size.width *sizeof(float), Size.height,
                                        hipMemcpyDeviceToDevice));

    // execute IDCT kernel
    //printf("Bind Quantization\n");
    checkCudaErrors(hipBindTextureToArray(TexSrc, Src));
    //printf("Run IDCT\n");
    CUDAkernel1IDCT<<< grid, threads >>>(Dst, (int) DstStride, 0, 0);
    checkCudaErrors(hipUnbindTexture(TexSrc));
    getLastCudaError("Kernel execution failed");
    //printf("Copy IDCT back\n");
    //copy quantized image block to host
    checkCudaErrors(hipMemcpy2D(ImgSrcF, ImgSrcFStride *sizeof(float),
                                 Dst, DstStride *sizeof(float),
                                 Size.width *sizeof(float), Size.height,
                                 hipMemcpyDeviceToHost));
    //printf("Convert to byte\n");
    //convert image back to byte representation
    AddFloatPlane(128.0f, ImgSrcF, ImgSrcFStride, Size);
    CopyFloat2Byte(ImgSrcF, ImgSrcFStride, ImgDst, Stride, Size);

    //clean up memory
    //printf("Cleanup\n");
    checkCudaErrors(hipFreeArray(Src));
    checkCudaErrors(hipFree(Dst));
    FreePlane(ImgSrcF);

    //return time taken by the operation
    return TimerCUDASpan;
  };

  float WrapperCUDA2(byte *ImgSrc, byte *ImgDst, int Stride, ROI Size){
    int StrideF;
    float *ImgF1 = MallocPlaneFloat(Size.width, Size.height, &StrideF);

    //convert source image to float representation
    CopyByte2Float(ImgSrc, Stride, ImgF1, StrideF, Size);
    AddFloatPlane(-128.0f, ImgF1, StrideF, Size);

    //allocate device memory
    float *src, *dst;
    size_t DeviceStride;
    checkCudaErrors(hipMallocPitch((void **)&src, &DeviceStride, Size.width * sizeof(float), Size.height));
    checkCudaErrors(hipMallocPitch((void **)&dst, &DeviceStride, Size.width * sizeof(float), Size.height));
    DeviceStride /= sizeof(float);

    //copy from host memory to device
    checkCudaErrors(hipMemcpy2D(src, DeviceStride * sizeof(float),
                                 ImgF1, StrideF * sizeof(float),
                                 Size.width * sizeof(float), Size.height,
                                 hipMemcpyHostToDevice));

    //create and start CUDA timer
    StopWatchInterface *timerCUDA = 0;
    sdkCreateTimer(&timerCUDA);

    //setup execution parameters
    dim3 GridFullWarps(Size.width / KER2_BLOCK_WIDTH, Size.height / KER2_BLOCK_HEIGHT, 1);
    dim3 ThreadsFullWarps(8, KER2_BLOCK_WIDTH/8, KER2_BLOCK_HEIGHT/8);

    //perform block-wise DCT processing and benchmarking
    const int numIterations = 100;

    for (int i = -1; i < numIterations; i++)
      {
        if (i == 0)
	  {
            checkCudaErrors(hipDeviceSynchronize());
            sdkResetTimer(&timerCUDA);
            sdkStartTimer(&timerCUDA);
	  }

        CUDAkernel2DCT<<<GridFullWarps, ThreadsFullWarps>>>(dst, src, (int)DeviceStride);
        getLastCudaError("Kernel execution failed");
      }

    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timerCUDA);

    //finalize timing of CUDA Kernels
    float avgTime = (float)sdkGetTimerValue(&timerCUDA) / (float)numIterations;
    sdkDeleteTimer(&timerCUDA);
    //printf("%f MPix/s //%f ms\n", (1E-6 * (float)Size.width * (float)Size.height) / (1E-3 * avgTime), avgTime);

    //setup execution parameters for quantization
    dim3 ThreadsSmallBlocks(BLOCK_SIZE, BLOCK_SIZE);
    dim3 GridSmallBlocks(Size.width / BLOCK_SIZE, Size.height / BLOCK_SIZE);

    // execute Quantization kernel
    CUDAkernelQuantizationFloat<<< GridSmallBlocks, ThreadsSmallBlocks >>>(dst, (int) DeviceStride);
    getLastCudaError("Kernel execution failed");

    //perform block-wise IDCT processing
    CUDAkernel2IDCT<<<GridFullWarps, ThreadsFullWarps >>>(src, dst, (int)DeviceStride);
    checkCudaErrors(hipDeviceSynchronize());
    getLastCudaError("Kernel execution failed");

    //copy quantized image block to host
    checkCudaErrors(hipMemcpy2D(ImgF1, StrideF *sizeof(float),
                                 src, DeviceStride *sizeof(float),
                                 Size.width *sizeof(float), Size.height,
                                 hipMemcpyDeviceToHost));

    //convert image back to byte representation
    AddFloatPlane(128.0f, ImgF1, StrideF, Size);
    CopyFloat2Byte(ImgF1, StrideF, ImgDst, Stride, Size);

    //clean up memory
    checkCudaErrors(hipFree(dst));
    checkCudaErrors(hipFree(src));
    FreePlane(ImgF1);

    //return time taken by the operation
    return avgTime;
  };

}
